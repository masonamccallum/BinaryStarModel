#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include "graphicsIncludes.h"
#include "vendorIncludes.h"
#include "Renderer.h"
#include "Grid.h"
#include "Camera.h"
#include "Input.h"
#include "GUI.h"
#include "Window.h"

//CUDA
#define N 1024 
#define BLOCK 256
#define DT 0.005
#define G 1.0
#define STOP_TIME 1.0
#define DAMP 1.0
#define H 1.0

float4 p[N];
float3 v[N], f[N];
float4 * p_GPU;
float3 * v_GPU,* f_GPU;
dim3 block; 
dim3 gridCUDA;


void set_initail_conditions(){
	int i,j,k,num,particles_per_side;
    float position_start, temp;
    float initail_seperation;

	temp = pow((float)N,1.0/3.0) + 0.99999;
	particles_per_side = temp;
	printf("\n cube root of N = %d \n", particles_per_side);
    position_start = -(particles_per_side -1.0)/2.0;
	initail_seperation = 1.0;
	
	for(i=0; i<N; i++)
	{
		p[i].w = 1.0;
	}
	
	num = 0;
	for(i=0; i<particles_per_side; i++)
	{
		for(j=0; j<particles_per_side; j++)
		{
			for(k=0; k<particles_per_side; k++)
			{
			    if(N <= num) break;
				p[num].x = position_start + i*initail_seperation;
				p[num].y = position_start + j*initail_seperation;
				p[num].z = position_start + k*initail_seperation;
				v[num].x = 0.0;
				v[num].y = 0.0;
				v[num].z = 0.0;
				num++;
			}
		}
	}
	
	block.x = BLOCK;
	block.y = 1;
	block.z = 1;
	
	gridCUDA.x = (N-1)/block.x + 1;
	gridCUDA.y = 1;
	gridCUDA.z = 1;
	
	hipMalloc( (void**)&p_GPU, N * sizeof(float4) );
	hipMalloc( (void**)&v_GPU, N * sizeof(float3) );
	hipMalloc( (void**)&f_GPU, N * sizeof(float3) );
}
__device__ float3 getBodyBodyForce(float4 p0, float4 p1)
{
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz;
    float r = sqrt(r2);
    
    float force  = (G*p0.w*p1.w)/(r2) - (H*p0.w*p1.w)/(r2*r2);
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

__global__ void getForces(float4 *pos, float3 *vel, float3 * force){
	int j,ii;
    float3 force_mag, forceSum;
    float4 posMe;
    __shared__ float4 shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	forceSum.z = 0.0;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	posMe.w = pos[id].w;
	    
    for(j=0; j < gridDim.x; j++){
    	shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    	__syncthreads();
   
		#pragma unroll 32
        for(int i=0; i < blockDim.x; i++){
        	ii = i + blockDim.x*j;
		    if(ii != id && ii < N){
		    	force_mag = getBodyBodyForce(posMe, shPos[i]);
			    forceSum.x += force_mag.x;
			    forceSum.y += force_mag.y;
			    forceSum.z += force_mag.z;
		    }
	   	 }
	}
	if(id < N){
	    force[id].x = forceSum.x;
	    force[id].y = forceSum.y;
	    force[id].z = forceSum.z;
    }
}
__global__ void moveBodies(float4* pos, float3* vel, float3* force)
{
    int id = threadIdx.x + blockDim.x*blockIdx.x;
	    vel[id].x += ((force[id].x-DAMP*vel[id].x)/pos[id].w)*DT;
	    vel[id].y += ((force[id].y-DAMP*vel[id].y)/pos[id].w)*DT;
	    vel[id].z += ((force[id].z-DAMP*vel[id].z)/pos[id].w)*DT;
	
	    pos[id].x += vel[id].x*DT;
	    pos[id].y += vel[id].y*DT;
	    pos[id].z += vel[id].z*DT;
}

void mouse_callback(GLFWwindow*, double, double);
// camera
Camera camera(glm::vec3(70.0f, 30.0f, 260.0f));
// timing
float deltaTime = 0.0f;	
float lastFrame = 0.0f;
const int width = 1600;
const int height = 1000;
float lastX = width/2.0f;
float lastY = height/2.0f;
bool firstMouse= true;


int main(void){

{
	float dt;
	float time = 0.0;
	float elapsedTime;	

	block.x = BLOCK;
	block.y = 1;
	block.z = 1;
	
	gridCUDA.x = (N-1)/block.x + 1;
	gridCUDA.y = 1;
	gridCUDA.z = 1;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	dt = DT;
    hipMemcpy( p_GPU, p, N * sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( v_GPU, v, N * sizeof(float3), hipMemcpyHostToDevice );
    
	while(time < STOP_TIME){	
		hipSetDevice(0);
		getForces<<<gridCUDA, block>>>(p_GPU, v_GPU, f_GPU);
		moveBodies<<<gridCUDA, block>>>(p_GPU, v_GPU, f_GPU);
		hipMemcpy(p , p_GPU, N * sizeof(float4), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		time += dt;
	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("\n\nGPU time = %3.1f milliseconds\n", elapsedTime);
}
	Window window(width, height, mouse_callback);

	glm::mat4 proj;
	proj = glm::perspective(glm::radians(45.0f), (float)width/(float)height, 0.1f, 400.0f);
	glm::mat4 view = camera.GetViewMatrix();
	glm::mat4 model = glm::mat4(1.0f);//glm::rotate(glm::mat4(1.0f), glm::radians(-100.0f), glm::vec3(1.0f,0.0f,0.0f));
	glm::mat4 mvp = proj * view * model;
	Renderer renderer;	
	Grid grid(5.0f, mvp);
	GUI gui(window.ptr);
	set_initail_conditions();

	
	//////////////////////////////////////////////////	
	//          CUDA INTEROP                        //
	//////////////////////////////////////////////////	
	VertexArray va;
	//float particles[N*4];
	//unsigned int index[N];

	//for(unsigned int i = 0; i < N; i++){
	//	index[i] = i;
	//}

	//for(int i = 0; i < 10; i++){
	//	for(int j = 0; j < 10; j++){
	//		for(int k = 0; k < 10; k++){
	//		}
	//	}
	//}
	
	float particles[16] = {
		0.0f,0.0f,0.0f,1.0f,
		10.0f,10.0f,10.0f,1.0f,
		10.0f,0.0f,100.0f,1.0f,
		0.0f,10.0f,0.0f,1.0f,
	};
	
	unsigned int index[4] = {
		0,1,2,3		
	};

	//for(int i=0; i < 20; i+=4){
	//	std::cout << "(" <<particles[i+0]<< "," <<particles[i+1]<< "," <<particles[i+2] << "," <<particles[i+3] << ")" << std::endl;
	//}
	
	VertexBuffer vb(4);
	VertexBufferLayout layout;		
	layout.Push<float>(4);
	va.AddBuffer(vb, layout);
	IndexBuffer ib(index, 4);//1000);
	Shader shader("../res/shaders/particle.shader");
	
	//hipGraphicsResource * resourceA;
	//VertexBuffer vb(100, true);
	//hipSetDevice(cutGetMaxGflopsDeviceId());
	//cudaGLSetGLDevice(cutGetMaxGflopsDeviceId());
	//hipGraphicsGLRegisterBuffer(...);
	//hipStream_t cuda_Stream;
	//hipStreamCreate(&cuda_Stream);
	//hipGraphicsMapResources(1, resource, cuda_Stream);
	//hipGraphicsUnmapResources(1, resource, cuda_Stream);
	//hipStreamDestroy(cuda_Stream);
	
    while(!window.shouldClose()){
        float currentFrame = glfwGetTime();
        deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		renderer.Clear();
		gui.NewFrame();
		Input::processInput(window.ptr, camera, deltaTime);

		glm::mat4 view = camera.GetViewMatrix();
		mvp = proj*view*model;
		//mvp = glm::mat4(1.0f);
		
		///////////////////////////////////////////
		//            CUDA INTEROP               //
		///////////////////////////////////////////
		

		shader.Bind();
		ib.Bind();
		shader.SetUniformMat4f("u_MVP", mvp);
		vb.Update(particles);
		renderer.Draw(va,ib,shader, GL_POINTS);
		///////////////////////////////////////////
		

		grid.Update(mvp);	
		renderer.Draw(grid);

		gui.CameraWindowUpdate(camera);
		gui.Render();
		window.Update();
    }
    return 0;
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos){
    if (firstMouse){
        lastX = xpos;
        lastY = ypos;
        firstMouse = false;
    }

    float xoffset = xpos - lastX;
    float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top
    lastX = xpos;
    lastY = ypos;

    camera.ProcessMouseMovement(xoffset, yoffset);
}