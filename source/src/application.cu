#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <cuda_gl_interop.h>
#include "graphicsIncludes.h"
#include "vendorIncludes.h"
#include "Renderer.h"
#include "Grid.h"
#include "Camera.h"
#include "Input.h"
#include "GUI.h"
#include "Window.h"

//CUDA
#define N 1024 
#define BLOCK 256
#define DT 0.0005
#define G 1.0
#define DAMP 1.0
#define H 0.00001

__device__ float3 getBodyBodyForce(float4 p0, float4 p1)
{
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz;
    float r = sqrt(r2);
    
    float force  = (G*p0.w*p1.w)/(r2) - (H*p0.w*p1.w)/(r2*r2);
    
    f.x = force*dx/r;
    f.y = force*dy/r;
    f.z = force*dz/r;
    
    return(f);
}

__global__ void getForces(float4 *pos, float3 *vel, float3 * force){
	int j,ii;
    float3 force_mag, forceSum;
    float4 posMe;
    __shared__ float4 shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	forceSum.z = 0.0;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	posMe.w = pos[id].w;
	    
    for(j=0; j < gridDim.x; j++){
    	shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    	__syncthreads();
   
		#pragma unroll 32
        for(int i=0; i < blockDim.x; i++){
        	ii = i + blockDim.x*j;
		    if(ii != id && ii < N){
		    	force_mag = getBodyBodyForce(posMe, shPos[i]);
			    forceSum.x += force_mag.x;
			    forceSum.y += force_mag.y;
			    forceSum.z += force_mag.z;
		    }
	   	 }
	}
	if(id < N){
	    force[id].x = forceSum.x;
	    force[id].y = forceSum.y;
	    force[id].z = forceSum.z;
    }
}
__global__ void moveBodies(float4* pos, float3* vel, float3* force)
{
    int id = threadIdx.x + blockDim.x*blockIdx.x;
	    vel[id].x += ((force[id].x-DAMP*vel[id].x)/pos[id].w)*DT;
	    vel[id].y += ((force[id].y-DAMP*vel[id].y)/pos[id].w)*DT;
	    vel[id].z += ((force[id].z-DAMP*vel[id].z)/pos[id].w)*DT;
	
	    pos[id].x += vel[id].x*DT;
	    pos[id].y += vel[id].y*DT;
	    pos[id].z += vel[id].z*DT;
}

// camera
Camera camera(glm::vec3(70.0f, 30.0f, 260.0f));
// timing
float deltaTime = 0.0f;	
float lastFrame = 0.0f;
const int width = 1600;
const int height = 1000;
float lastX = width/2.0f;
float lastY = height/2.0f;
bool firstMouse= true;

void mouse_callback(GLFWwindow* window, double xpos, double ypos){
    if (firstMouse){
        lastX = xpos;
        lastY = ypos;
        firstMouse = false;
    }

    float xoffset = xpos - lastX;
    float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top
    lastX = xpos;
    lastY = ypos;

    camera.ProcessMouseMovement(xoffset, yoffset);
}


int main(void){
	Window window(width, height, mouse_callback);

	glm::mat4 proj;
	proj = glm::perspective(glm::radians(45.0f), (float)width/(float)height, 0.1f, 400.0f);
	glm::mat4 view = camera.GetViewMatrix();
	glm::mat4 model = glm::mat4(1.0f);//glm::rotate(glm::mat4(1.0f), glm::radians(-100.0f), glm::vec3(1.0f,0.0f,0.0f));
	glm::mat4 mvp = proj * view * model;
	Renderer renderer;	
	Grid grid(5.0f, mvp);
	GUI gui(window.ptr);
	//set_initail_conditions();

	
	//////////////////////////////////////////////////	
	//          CUDA INTEROP                        //
	//////////////////////////////////////////////////	
	float particles_CPU[N*4];
	
	unsigned int index[N];

	for(unsigned int i = 0; i < N; i++){
		index[i] = i;
	}
	
	float point[4];
	int idx = 0;
	for(int i = 0; i < 10; i++){
		for(int j = 0; j < 10; j++){
			for(int k = 0; k < 10; k++){
				point[0] = i * 10; point[1] = j * 10; point[2] = k * 10; point[3] = 1.0f;
				for(int m =0; m < 4; m++){
					particles_CPU[idx*4 + m] = point[m];
				}
				idx++;
			}
		}
	}
	
	VertexArray va;
	VertexBuffer vb(1024);
	VertexBufferLayout layout;		
	layout.Push<float>(4);
	va.AddBuffer(vb, layout);
	IndexBuffer ib(index, 1000);
	Shader shader("../res/shaders/particle.shader");
	
	float vel_CPU[N*3] = {0};
	float force_CPU[N*3] = {0};
	float4* posGPU;
	float3 *velGPU, *forceGPU;
	hipMalloc((void**)&posGPU, 1024*sizeof(float4));
	hipMalloc((void**)&velGPU, 1024*sizeof(float3));
	hipMalloc((void**)&forceGPU, 1024*sizeof(float3));
	hipMemcpy(forceGPU, force_CPU, 1024*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(velGPU,vel_CPU, 1024*sizeof(float3), hipMemcpyHostToDevice);
	
    while(!window.shouldClose()){
        float currentFrame = glfwGetTime();
        deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		renderer.Clear();
		gui.NewFrame();
		Input::processInput(window.ptr, camera, deltaTime);

		glm::mat4 view = camera.GetViewMatrix();
		mvp = proj*view*model;

		
		hipMemcpy(posGPU, particles_CPU, 1024*sizeof(float4), hipMemcpyHostToDevice);
		getForces<<<1024, 1>>>(posGPU, velGPU, forceGPU);
		moveBodies<<<1024, 1>>>(posGPU, velGPU, forceGPU);
		hipMemcpy(particles_CPU, posGPU, 1024*sizeof(float4), hipMemcpyDeviceToHost);

		vb.Update(particles_CPU);
		ib.Bind();
		shader.Bind();
		shader.SetUniformMat4f("u_MVP", mvp);
		renderer.Draw(va,ib,shader,GL_POINTS);
		
		grid.Update(mvp);	
		renderer.Draw(grid);

		gui.CameraWindowUpdate(camera);
		gui.Render();
		window.Update();
    }
    return 0;
}